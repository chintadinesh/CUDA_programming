
#include <hip/hip_runtime.h>
#include <vector>
#include <cassert>
#include <iostream>

__global__ void square(float *d_out, float *d_in) {
  d_out[threadIdx.x] = d_in[threadIdx.x] * d_in[threadIdx.x];
}

int main() {
  constexpr int ARRAY_SIZE = 64;
  constexpr int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  // host side memory allocation
  std::vector<float> h_in(ARRAY_SIZE, 0);
  for(int i = 0; i < ARRAY_SIZE; ++i) h_in[i] = static_cast<float>(i);
  std::vector<float> h_out(ARRAY_SIZE, 0);

  // GPU memory
  float *d_in, *d_out;
  hipMalloc((void **)&d_in, ARRAY_BYTES);
  hipMalloc((void **)&d_out, ARRAY_BYTES);

  hipMemcpy(d_in, &h_in[0], ARRAY_BYTES, hipMemcpyHostToDevice);

  square<<<1, ARRAY_SIZE>>>(d_out, d_in);

  hipMemcpy(&h_out[0], d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

  // clean up
  hipFree(d_in);
  hipFree(d_out);


  // verify the results
  for(int i = 0; i < ARRAY_SIZE; i++) assert(h_out[i] == h_in[i] * h_in[i]);
  std::cout << "TEST PASSED!\n";

  return 0; }